#include "hip/hip_runtime.h"
/*
����O���t�T���𗘗p�����o�[�W����
�����W���^ - �v���Z�b�T�ԂŃm�[�h��open list�����L��������ɋ��L����
http://d.hatena.ne.jp/hanecci/20110205/1296924411
*/

#include <hip/hip_runtime.h>
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>       // helper functions for CUDA error checking and initialization
#include <hip/hip_runtime.h>
#include <memory>
#include <iostream>
#include <cassert>
#include <algorithm>
#include <stdio.h>
#include <time.h>
#include <vector>
#include <string>
#include <sstream>
#include <list>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "Timer.h"
#include "CpuGpuData.cuh"
#include <time.h>


/*
############################ Warning #####################
GPU�v���O���~���O�ł͉ϒ��z����g�������Ȃ����ߒ萔�l�𗘗p���Ă��܂��B
�K�X�l��ς��邱��
########################################################
*/

#define MAX3(a,b,c) ((a<b)? ((b<c)? c: b):  ((a<c)? c: a))

#define CLUSTER_NUM 3 /*�N���X�^��*/
#define DATA_NUM 150 /*�f�[�^��*/
#define TEMP_SCENARIO_NUM 20 /*���x�J�ڃV�i���I�̐�*/
#define P 4 /* ������ */
#define EPSIRON 0.001 /* ���e�G���[*/
#define N 128 /* �f�[�^�Z�b�g�� */

typedef unsigned  int uint;
using namespace std;

/*
�f�o�C�X�ɓn������/�󂯎��̃f�[�^�Z�b�g
device_vector�ɓ˂����ލ\���̂̒��͂ǂ����ʏ�̔z��ŗǂ��炵���B
���ׁ̈A�ϒ��z��͎g�p�ł��Ȃ��\���������B
FCM�ł�dik, uik...
*/
typedef struct{
	float dik[DATA_NUM*CLUSTER_NUM];
	float uik[DATA_NUM*CLUSTER_NUM];
	float xk[DATA_NUM*P];
	float vi[CLUSTER_NUM*P];
	float vi_bak[CLUSTER_NUM*P];			//���ꉷ�x�ł̑O��vi
	float Vi_bak[CLUSTER_NUM*P];			//�قȂ鉷�x�ł̑O��vi
	int error;	//	�G���[��
	float T[TEMP_SCENARIO_NUM]; //	���x�J�ڂ̃V�i���I
	int results[DATA_NUM];	//	���s����
	float q;		//	q�l
	int t_pos;		//	���x�V�i���I�Q�ƈʒu
	int t_change_num;	//	���x�ύX��
	float jfcm;
	BOOL is_finished; //�N���X�^�����O�I�������𖞂��������ǂ���
}DataSet;





__global__ void device_FCM(DataSet *ds);
__device__ void __device_calc_convergence(float *vi, float *vi_bak, int iSize, int pSize, float *err);
__device__ void __device_VFA(float *, float, int, float, float);
__device__ void __device_update_vi(float *uik, float *xk, float *vi, float iSize, int kSize, int pSize, float m);
__device__ void __device_update_uik(float *, float *, int, int, float);
__device__ void __device_update_uik_with_T(float *uik, float *dik, int iSize, int kSize, float q, float T);
__device__ void __device_distance(float *, float *, float *, int);
__device__ void __device_update_dik(float *dik, float *vi, float *xk, int iSize, int kSize, int pSize);
__device__ void __device_jfcm(float *uik, float *dik, float *jfcm, float m, int iSize, int kSize);
__device__ void __device_jtsallis(float *uik, float *dik, float *jfcm, float q, float T, int iSize, int kSize);
__device__ void __device_eval(float *uik, int *results, int iSize, int kSize);
__device__ void __device_iris_error(float *uik, int *error, int iSize, int kSize);

float my_random(float min, float max){
	return min + (float)(rand() * (max - min) / RAND_MAX);
}

void deepcopy_vector(vector<int> *src, vector<int> *dst){
	for (int i = 0; i < src->size(); i++){
		(*dst)[i] = (*src)[i];
	}	
}

void listT_to_str(std::stringstream ss, float *T, int size){
	for (int i = 0; i < size; i++){
		ss << T[i];
	}
}

void init_datasets(DataSet ds[]){
	float tmp_xk[DATA_NUM*P];
	for (int k = 0; k < DATA_NUM / 2; k++){
		tmp_xk[k * P + 0] = my_random(0.0, 0.5);
		tmp_xk[k * P + 1] = my_random(0.0, 0.5);
	}
	for (int k = DATA_NUM / 2; k < DATA_NUM; k++){
		tmp_xk[k * P + 0] = my_random(0.5, 1.0);
		tmp_xk[k * P + 1] = my_random(0.5, 1.0);
	}

	for (int j = 0; j < N; j++){
		ds[j].t_pos = 0;
		ds[j].q = 5.0;		//	�Ƃ肠����q��2.0�Œ�
		ds[j].T[0] = pow(20.0f, (j + 1.0f - N/2.0f) / (N/2.0f));  // Thigh�ŏ������x������
		ds[j].is_finished = FALSE;
		for (int i = 0; i < CLUSTER_NUM; i++){
			//	�����_��������
			ds[j].vi[i * P + 0] = (double)rand() / RAND_MAX;
			ds[j].vi[i * P + 1] = (double)rand() / RAND_MAX;
		}
		for (int k = 0; k < DATA_NUM; k++){
			ds[j].xk[k * P + 0] = tmp_xk[k * P + 0];
			ds[j].xk[k * P + 1] = tmp_xk[k * P + 1];
		}

	}


	/*
	for (int k = 0; k < DATA_NUM / 2; k++){
	ds->xk[k * P + 0] = my_random(0.0, 0.5);
	ds->xk[k * P + 1] = my_random(0.0, 0.5);
	//h_ds[0].xk[k * P + 0] = my_random(0.0, 5.0);
	//h_ds[0].xk[k * P + 1] = my_random(0.0, 0.5);
	}
	for (int k = DATA_NUM / 2; k < DATA_NUM; k++){
	//[0].xk[k * P + 0] = my_random(0.75, 1.0);
	//h_ds[0].xk[k * P + 1] = my_random(0.75, 1.0);
	ds->xk[k * P + 0] = my_random(0.5, 1.0);
	ds->xk[k * P + 1] = my_random(0.5, 1.0);
	}
	*/
}

void iris_datasets(DataSet ds[]){
	FILE *fp = fopen("data/iris.txt", "r");
	float tmp_xk[DATA_NUM*P];
	for (int k = 0; k < DATA_NUM; k++){
		for (int p = 0; p < P; p++){
			float tmp;
			fscanf(fp, "%f", &tmp);
			tmp_xk[k * P + p] = tmp;
		}
	}
	for (int j = 0; j < N; j++){
		ds[j].t_pos = 0;
		ds[j].q = 2.0;		//	�Ƃ肠����q��2.0�Œ�
		ds[j].T[0] = pow(20.0f, (j + 1.0f - N / 2.0f) / (N / 2.0f));  // Thigh�ŏ������x������
		ds[j].is_finished = FALSE;
		for (int i = 0; i < CLUSTER_NUM; i++){
			for (int p = 0; p < P; p++){
				ds[j].vi[i * P + p] = my_random(0.0, 10.0);
			}
		}
		for (int k = 0; k < DATA_NUM; k++){
			for (int p = 0; p < P; p++){
				ds[j].xk[k * P + p] = tmp_xk[k * P + p];
			}
		}
	}
	fclose(fp);
}

void print_result(const DataSet *ds){
	printf("T=");
	for (int i = 0; i < TEMP_SCENARIO_NUM && ds->T[i]!=0.0; i++) printf("%1.2f ", ds->T[i]);
	//printf("\n");
	//printf("q=%f", ds->q);
	//printf("\n");

	/*
	printf("results=\n");
	for (int i = 0; i < DATA_NUM; i++){
			printf("%d ", ds->results[i]);
			if ((i + 1) % 20 == 0) printf("\n");
	}
	printf("\n");
	*/

	printf("error=%d\n", ds->error);
	//printf("jfcm = %f\n", ds->jfcm);

	/*
	printf("vi_bak=");
	for (int i = 0; i < CLUSTER_NUM; i++){
	for (int p = 0; p < P; p++){
	printf("%1.2f ", ds->vi_bak[i*P + p]);
	}
	}
	printf("\n");
	*/
}

int compare(const int *target, int *sample, int size){

	//	[0,1,2]�̑g�ݍ��킹�̍쐬�p�z��Ɛ����p�^�[��
	vector<int> pattern = vector<int>();
	vector<int> good_pattern = vector<int>();
	for (int i = 0; i < 3; i++){
		pattern.push_back(i);
		good_pattern.push_back(0);
	}

	//	�G���[�ŏ��l
	int min_error = INT_MAX;

	//	���ׂĂ̒u���p�^�[���Ń}�b�`���O
	do{
		//	�G���[��
		int error = 0;
		//	���ׂẴf�[�^�ɂ��āA
		for (int j = 0; j < size; j++){
			if (0 <= sample[j] && sample[j] < 3){
				int index = pattern[sample[j]];	//	�u������
				if (target[j] != index) error++;	//	���������
			}
			else{
				error++;
			}
		}
		//	�땪�ސ������Ȃ���Γ���ւ���
		if (error < min_error){
			min_error = error;
			deepcopy_vector(&pattern, &good_pattern);
		}

	} while (next_permutation(pattern.begin(), pattern.end()));

	//	�u���p�^�[���𗘗p���āA�C���f�b�N�X��u������
	for (int i = 0; i < size; i++){
		if (0 <= sample[i] && sample[i] < 3){
			sample[i] = good_pattern[sample[i]];
		}
	}
	return min_error;
}


int main(){
	srand((unsigned)time(NULL));


	/*
	�z�X�g�ƃf�o�C�X�̃f�[�^�̈���m�ۂ���
	DataSetIn, DataSetOut��FCM�ɗp����f�[�^�̏W���A�\���̂Ȃ̂ŁA�q�m�[�h�����m�ۂ���΂悢
	�m�ې�1�ɂ���ƕ��񉻂��s�킸�A�ʏ�VFA+FCM�ōs��
	*/
	thrust::device_vector<DataSet> d_ds(N);
	thrust::host_vector<DataSet> h_ds(N);

	/*
	������Ԃ��쐬����
	TODO:�����_���p�^�[��, �����̃f�[�^�Z�b�g�p�^�[����p�ӂ���K�v����
	*/
	//const float listT[N] = { 50.0, 20.0, 10.0, 5.0, 2.0, 1.0};
	init_datasets(&h_ds[0]);

	/*
	��������BFS�œW�J����
	*/
	for (int i = 0; i < 20; i++){

		/*
		HOST����������GPU�������փR�s�[
		*/
		d_ds = h_ds;

		/*
		DataSetIn�ɑ΂�FCM�@��K�p���邱�Ƃɂ��ADataSetOut���擾����
		*/
		device_FCM << <1, N>> >(thrust::raw_pointer_cast(d_ds.data()));
		hipDeviceSynchronize();

		hipError_t error = hipGetLastError();
		if (error != hipSuccess) {
			fprintf(stderr, "%s\n", hipGetErrorString(error));
		}

		/*
		GPU����������HOST�������փR�s�[
		�C�R�[���ő���ł���炵��
		*/
		h_ds = d_ds;

	}



	/*
		�����쐬
	*/
	int targets[150];
	for (int i = 0; i < 50; i++) targets[i] = 0;
	for (int i = 50; i < 100; i++) targets[i] = 1;
	for (int i = 100; i < 150; i++) targets[i] = 2;
	for (int i = 0; i < N; i++){
		/*
		for (int j = 0; j < DATA_NUM; j++){
			printf("%d ", h_ds[i].results[j]);
		}
		printf("\n");
		*/
		h_ds[i].error = compare(targets, h_ds[i].results, DATA_NUM);
	}

	/*
		uik���t�@�C���ɏ�������
	*/
	for (int n = 0; n < N; n++){
		char buf[256];
		sprintf(buf, "out/uik%d.txt", n);
		FILE *fp2 = fopen(buf, "w");
		for (int k = 0; k < DATA_NUM; k++){
			for (int i = 0; i < CLUSTER_NUM; i++){
				fprintf(fp2, "%f ", h_ds[n].uik[i*DATA_NUM + k]);
			}
			fprintf(fp2, "\n");
		}
		fclose(fp2);
	}

	/* results���������� */
	for (int n = 0; n < N; n++){
		char buf[256];
		sprintf(buf, "out/results%d.txt", n);
		FILE *fp3 = fopen(buf, "w");
		for (int i = 0; i < DATA_NUM; i++){
			fprintf(fp3, "%d ", h_ds[n].results[i]);
		}
		fclose(fp3);
	}


	/*
		���ʂ�\������
	*/
	printf("--------------------The Clustering Result----------------------\n");
	for (int i = 0; i < N; i++){
			printf("[%d] ", i);
			print_result(&h_ds[i]);
	}



	/*
	hipError_t err = hipDeviceReset();
	hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "%s\n", hipGetErrorString(err));
	}
	*/

	return 0;
}






/*
FCM��������֐�
GPU��Vi�̎���������s��
*/
__device__ void __device_calc_convergence(float *vi, float *vi_bak, int iSize, int pSize, float *err){
	float max_error = 0;
	for (int i = 0; i < iSize; i++){
		float sum = 0.0;				//	�N���X�^���S�̈ړ��ʂ��v�Z����
		for (int p = 0; p < pSize; p++){
			sum += pow(vi[i*pSize + p] - vi_bak[i*pSize + p], 2.0f);
		}
		max_error = MAX(max_error, sum);	//	�ł��傫���ړ��ʂ𔻒f��ɂ���
	}
	*err = max_error;
}

/*
FCM��p�֐�
VFA�ŉ��x��������
*/
__device__ void __device_VFA(float *T, float Thigh, int k, float D, float Cd = 2.0){
	*T = Thigh * exp(-Cd*pow((float)k - 1, 1.0f / D));
}

/*
FCM
�N���X�^���S���X�V
*/
__device__  void __device_update_vi(float *uik, float *xk, float *vi, int iSize, int kSize, int pSize, float m){
	for (int i = 0; i < iSize; i++){
		//	�Ɨ����Ă��邽�߁A����ɗ��p���鍇�v�l���o���Ă���
		float sum_down = 0;
		for (int k = 0; k < kSize; k++){
			sum_down += pow(uik[i*kSize + k], m);
		}
		//	���q���v�Z����	
		for (int p = 0; p < pSize; p++){
			float sum_up = 0;
			for (int k = 0; k < kSize; k++){
				sum_up += pow(uik[i*kSize + k], m) * xk[k*pSize + p];
			}
			vi[i*pSize + p] = sum_up / sum_down;
		}
	}
}

/*
FCM
uik���X�V����
*/
__device__ void __device_update_uik(float *uik, float *dik, int iSize, int kSize, float m){
	for (int i = 0; i < iSize; i++){
		for (int k = 0; k < kSize; k++){
			float sum = 0;
			for (int j = 0; j < iSize; j++){
				sum += pow((float)(dik[i*kSize + k] / dik[j*kSize + k]), float(1.0 / (m - 1.0)));
			}
			uik[i*kSize + k] = 1.0 / sum;
		}
	}
}

/*
�A�j�[�����O��uik���X�V����
*/
__device__ void __device_update_uik_with_T(float *uik, float *dik, int iSize, int kSize, float q, float T){
	for (int i = 0; i < iSize; i++){
		for (int k = 0; k < kSize; k++){
			float sum = 0;
			for (int j = 0; j < iSize; j++){
				sum += pow((1.0f - (1.0f / T)*(1.0f - q)*dik[j*kSize + k]), 1.0f / (1.0f - q));
			}
			float up = pow((1.0f - (1.0f / T)*(1.0f - q)*dik[i*kSize + k]), 1.0f / (1.0f - q));
			uik[i*kSize + k] = up / sum;
		}
	}
}

/*
eval
*/
__device__ void __device_eval(float *uik, int *results, int iSize, int kSize){
	for (int k = 0; k < kSize; k++){
		results[k] = 0;
		float maxValue = uik[0*kSize + k];
		for (int i = 1; i < iSize; i++){
			if (maxValue < uik[i*kSize + k]){
				maxValue = uik[i*kSize + k];
				results[k] = i;
			}
		}
	}

}

/*
	������IRIS�̃f�[�^�Ɣ�r���Ă����Ԉ���Ă��邩�擾����
	50x3�Ƃ���
	�b��I�ȏ��u�ł�
*/
__device__ void __device_iris_error(float *uik, int *error, int iSize, int kSize){
	int sum[] = { 0, 0, 0 };
	int err = 0;
	
	for (int k = 0; k < kSize; k++){
		float maxValue = uik[0*kSize +k];
		int maxIndex = 0;
		for (int i = 1; i < iSize; i++){
			//	�ł��傫��index���擾
			float value = uik[i*kSize + k];
			if (maxValue < value){
				value = maxValue;
				maxIndex = i;
			}
		}
		//	�傫��index�ɍ��v�l�𑫂�
		sum[maxIndex] ++;
		
		//	50�ɂȂ�����G���[���v�Z����
		if (k == 49 || k == 99 || k == 149){
			err += 50 - MAX3(sum[0], sum[1], sum[2]);
			for (int m = 0; m <  3; m++) sum[m] = 0;
		}	
		
	}
	*error = err;

}


/*
�ړI�֐�JFCM���`���Ă���
�œK���̔��f�ɗ��p����
*/
__device__ void __device_jfcm(float *uik, float *dik, float *jfcm, float m, int iSize, int kSize){
	float total = 0.0;
	for (int i = 0; i < iSize; i++){
		for (int k = 0; k < kSize; k++){
			total += pow(uik[i*kSize + k], 1.0f) * dik[i*kSize + k];
		}
	}
	*jfcm = total;
}

__device__ void __device_jtsallis(float *uik, float *dik, float *j, float q, float T, int iSize, int kSize){
	float total = 0.0;
	for (int i = 0; i < iSize; i++){
		for (int k = 0; k < kSize; k++){
			float ln_q = (pow(uik[i*kSize + k], 1.0f - q) - 1.0f) / (1.0f - q);
			total += pow(uik[i*kSize + k], q) * dik[i*kSize + k] + T * pow(uik[i*kSize + k], q) *ln_q;
		}
	}
	*j = total;
}

/*
FCM
dik���X�V����
*/
__device__ void __device_update_dik(float *dik, float *vi, float *xk, int iSize, int kSize, int pSize){
	for (int i = 0; i < iSize; i++){
		for (int k = 0; k < kSize; k++){
			float sum = 0.0;
			for (int p = 0; p < pSize; p++){
				sum += pow(float(xk[k*pSize + p] - vi[i*pSize + p]), 2.0f);
			}
			//	dik->setValue(k, i, sqrt(sum));
			//dik[k*iSize + i] = sum;
			dik[i*kSize + k] = sum;
		}
	}
}


/*
FCM
�����𑪂�
*/
__device__ void __device_distance(float* d, float *v1, float *v2, int pSize){
	int p;
	double total = 0.0;
	for (p = 0; p < pSize; p++){
		/* v1[p] * v2[p] */
		/* 1�����z��Ŋm�ۂ���Ă���ꍇ�ɔ����Ă����Ă������Ă��܂� */
		total += pow(*(v1 + p) - *(v2 + p), 2);
	}
	*d = total;
}

/*
array���R�s�[����
*/
__device__ void __device_copy_float(float *src, float *dst, int size){
	for (int i = 0; i < size; i++){
		dst[i] = src[i];
	}
}

/*
	FCM
*/
__global__ void device_FCM(DataSet *ds){
	int i = threadIdx.x;
	float err;
	float t;
	float jfcm;

	//	�N���X�^�����O���Ȃ�
	if (ds[i].is_finished){
		return;
	}

	//	uik���X�V����
	__device_update_dik(ds[i].dik, ds[i].vi, ds[i].xk, CLUSTER_NUM, DATA_NUM, P);
	__device_update_uik_with_T(ds[i].uik, ds[i].dik, CLUSTER_NUM, DATA_NUM, ds[i].q, ds[i].T[ds[i].t_pos]);

	//	���ł�jfcm�����߂Ă���
	__device_jtsallis(ds[i].uik, ds[i].dik, &jfcm, ds[i].q, ds[i].T[ds[i].t_pos], CLUSTER_NUM, DATA_NUM);
	ds[i].jfcm = jfcm;

	//	vi�̃o�b�N�A�b�v�����
	__device_copy_float(ds[i].vi, ds[i].vi_bak, CLUSTER_NUM*P);

	//	vi(centroids)���X�V����
	__device_update_vi(ds[i].uik, ds[i].xk, ds[i].vi, CLUSTER_NUM, DATA_NUM, P, ds[i].q);

	//	���ꉷ�x�ł̎����𔻒�
	//	�������Ă��Ȃ���΂��̂܂܂̉��x�ŌJ��Ԃ�
	__device_calc_convergence(ds[i].vi, ds[i].vi_bak, CLUSTER_NUM, P, &err);
	//err= 0; // ���x��������
	if (EPSIRON < err){
		//	���x���������Ɋ֐����I��
		ds[i].t_pos++;
		ds[i].T[ds[i].t_pos] = ds[i].T[ds[i].t_pos - 1];
		return;
	}

	//	�O�̉��x�Ƃ̎����𔻒�
	//	�������Ă�����I��
	__device_calc_convergence(ds[i].vi, ds[i].Vi_bak, CLUSTER_NUM, P, &err);
	//err = 0; // �I��
	if (err < EPSIRON){
		//	���̎��_�ŃN���X�^�����O���I������
		ds[i].is_finished = TRUE;
		__device_eval(ds[i].uik, ds[i].results, CLUSTER_NUM, DATA_NUM);
		//int cnt;
		//__device_iris_error(ds[i].uik, &cnt, CLUSTER_NUM, DATA_NUM);
		//ds[i].error = cnt;
		return;
	}

	//	�o�b�N�A�b�v
	//	���x��������O��vi��ۑ�
	__device_copy_float(ds[i].vi, ds[i].Vi_bak, CLUSTER_NUM*P);

	// �������Ă��Ȃ���Ή��x�������ČJ��Ԃ�
	ds[i].t_pos++; 
	ds[i].t_change_num++;
	__device_VFA(&t, ds[i].T[0], ds[i].t_change_num + 1, P);
	ds[i].T[ds[i].t_pos] = t;



}



/*
�֐�node_expand()
�J�ڐ�̉��x�����肵�A�q���𐶐�����B
�����ł�FCM�@�̎��s�͂����A�e����l�������p���̂݁B
node_execute()��FCM�@��1�񂾂����s����B
TODO: ��������q���̐�, ���񉷓x�̌���B
*/
/*
void node_expand(const node_t *node, std::vector<node_t> *children){
if (node->temp_scenario.size() > 2) return;

for (int i = 0; i < 3; i++){
node_t child;
std::copy(node->temp_scenario.begin(), node->temp_scenario.end(), std::back_inserter(child.temp_scenario));
child.temp_scenario.push_back(node->temp_scenario.back() / 2.0f);
children->push_back(child);
}

}
*/



/*
�m�[�h��GPU�œW�J����
*/
__global__ void gpu_node_execute(int *results){
	int idx = threadIdx.x;
	results[idx] = threadIdx.x;
}


/*
���D��T��(Breadth First Search)
*/
/*
int BFS(node_t node){
int n = 0; // ����܂łɒT�������m�[�h��
std::list<node_t> open_list;	//	�I�[�v�����X�g

open_list.push_back(node);
while (!open_list.empty()){
node = open_list.front();
for (int i = 0; i<node.temp_scenario.size(); i++){
printf("%f ", node.temp_scenario[i]);
}
printf("\n");

if (node_is_goal(&node)){
return n;
}

n++;
open_list.pop_front();

//	CPU�Ŏq�m�[�h��W�J����
std::vector<node_t> children;
node_expand(&node, &children);

//	CPU��GPU�f�[�^�R�s�[
//	node_t�^�̂܂܂ł͗��p�ł��Ȃ��̂ŕϊ����Ă���
thrust::device_vector<int> d_results(8);

for (auto it = children.begin(); it != children.end(); it++){

}

//	�����FCM���s����
gpu_node_execute << <1, 8 >> >(thrust::raw_pointer_cast(d_results.data()));

//	CPU��GPU�f�[�^�R�s�[
// node_t�^�ɕϊ����Ă���
//auto it_results = d_results.begin
for (auto it = children.begin(); it != children.end(); it++){
(*it).result.push_back(0);
}

//	open_list�ɒǉ����čēx�T��
int n = children.size();
for (int i = 0; i < n; i++){
open_list.push_back(children[i]);
}

}
return -1;
}
*/

